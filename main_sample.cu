#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <thrust/device_vector.h>
#include "csrmatrix.h"
#include "sparsematrix.h"
#include "ConjugateGradient.h"


int main()
{
      /**********************************/
      /********** 入力値の準備 **********/
      /**********************************/
      long N = 1024;
      sparse::matrix<double> A(N);

	// 中央差分行列を準備する
	//（対角項が2でその隣が1になる、↓こんなやつ）
	// | 2 1 0 0 0 0 0 0 ・・・ 0 0 0|
	// | 1 2 1 0 0 0 0 0 ・・・ 0 0 0|
	// | 0 1 2 1 0 0 0 0 ・・・ 0 0 0|
	// | 0 0 1 2 1 0 0 0 ・・・ 0 0 0|
	// | 0 0 0 1 2 1 0 0 ・・・ 0 0 0|
	// | 0 0 0 0 1 2 1 0 ・・・ 0 0 0|
	// | 0 0 0 0 0 1 2 1 ・・・ 0 0 0|
	// | 0 0 0 0 0 0 1 2 ・・・ 0 0 0|
	// | 0 0 0 0 0 0 0 0 ・・・ 2 1 0|
	// | 0 0 0 0 0 0 0 0 ・・・ 1 2 1|
	// | 0 0 0 0 0 0 0 0 ・・・ 0 1 2|

      for (long i = 0; i < N; i++) {
	  A[i][i] = 2;
	  if(i > 0) A[i][i-1] = 1;
	  if(i < N-1) A[i][i+1] = 1;
      }

      // 解ベクトル(初期値)の設定
      double* x = new double[N];
      for (long i = 0; i < N; i++) x[i] = 1.0;

      // 右辺ベクトルを設定 (b==0ならばbを自動生成)
      double* b = new double[N];
      for (long i = 0; i < N; i++) b[i] = 0.0;

      ConjugateGradient(A,x,b);
      for (long i = 0; i < N; i++) cout << x[i] << endl;
      delete[] x;
      delete[] b;
      return 0;
}
