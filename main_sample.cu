#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <thrust/device_vector.h>
#include "simple.h"
#include "custo.h"
using namespace std;
using namespace thrust;
using namespace custo;
#include "crs.h"
#include "d.h"
#include "operator.h"
#include "ConjugateGradient.h"

int main()
{
      /**********************************/
      /********** 入力値の準備 **********/
      /**********************************/
      long i, n = 1024;
      matrix<double> A(n);
      device_vector<double> x(n), b(n);

        // 中央差分行列を準備する
	//（対角項が2でその隣が1になる、↓こんなやつ）
	// | 2 1 0 0 0 0 0 0 ・・・ 0 0 0|
	// | 1 2 1 0 0 0 0 0 ・・・ 0 0 0|
	// | 0 1 2 1 0 0 0 0 ・・・ 0 0 0|
	// | 0 0 1 2 1 0 0 0 ・・・ 0 0 0|
	// | 0 0 0 1 2 1 0 0 ・・・ 0 0 0|
	// | 0 0 0 0 1 2 1 0 ・・・ 0 0 0|
	// | 0 0 0 0 0 1 2 1 ・・・ 0 0 0|
	// | 0 0 0 0 0 0 1 2 ・・・ 0 0 0|
	// | 0 0 0 0 0 0 0 0 ・・・ 2 1 0|
	// | 0 0 0 0 0 0 0 0 ・・・ 1 2 1|
	// | 0 0 0 0 0 0 0 0 ・・・ 0 1 2|

      for (i = 0; i < n; i++) {
	  A[i][i] = 2;
	  if(i > 0) A[i][i-1] = 1;
	  if(i < n-1) A[i][i+1] = 1;
      }

      // 解ベクトル(初期値)の設定
      for (i = 0; i < n; i++) x[i] = 1.0;

      // 右辺ベクトルを設定 (b==0ならばbを自動生成)
      for (i = 0; i < n; i++) b[i] = 0.0;

      ConjugateGradient(A,x,b);

      for (i = 0; i < n; i++) cout << x[i] << endl;
      return 0;
}
